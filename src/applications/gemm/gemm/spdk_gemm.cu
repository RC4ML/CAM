#include "lightbam.cuh"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hipblas.h>
#include <algorithm>
#include "gemm.cuh"
#include <vector>
#include "spdk_read.h"

typedef float fp_t;
int main(int argc, char *argv[])
{
    if (argc != 10)
    {
        printf("Usage: %s m n k a_offset b_offset c_offset block_size max_io_size num_ssds\n", argv[0]);
        return 1;
    }
    int m = parse_offset(argv[1]);
    int n = parse_offset(argv[2]);
    int k = parse_offset(argv[3]);
    uint64_t a_offset = parse_offset(argv[4]);
    uint64_t b_offset = parse_offset(argv[5]);
    uint64_t c_offset = parse_offset(argv[6]);
    uint64_t block_size = parse_offset(argv[7]);
    uint64_t max_io_size = parse_offset(argv[8]);
    int num_ssds = atoi(argv[9]);
    if (m % block_size != 0 || n % block_size != 0)
    {
        std::cout<<"m and n must be a multiple of block_size"<<std::endl;
        return 1;
    }
    int m_blocks = m / block_size;
    int n_blocks = n / block_size;
    if (block_size * sizeof(fp_t) % max_io_size != 0)
    {
        std::cout<<"block_size * sizeof(fp_t) must be a multiple of max_io_size"<<std::endl;
        return 1;
    }
    int num_queues_per_ssd = CEIL(block_size * k * sizeof(fp_t), num_ssds * 4096 * max_io_size) + 1;
    fp_t *a0, *a1, *b0, *b1, *c0, *c1;
    printf("max_io_size = %ld\n", max_io_size); 
    cam_init(max_io_size);
    // a0 = (fp_t*)alloc_gpu(block_size * k * sizeof(fp_t));
    // a1 = (fp_t*)alloc_gpu(block_size * k * sizeof(fp_t));
    // b0 = (fp_t*)alloc_gpu(block_size * k * sizeof(fp_t));
    // b1 = (fp_t*)alloc_gpu(block_size * k * sizeof(fp_t));
    // c0 = (fp_t*)alloc_gpu(block_size * block_size * sizeof(fp_t));
    // c1 = (fp_t*)alloc_gpu(block_size * block_size * sizeof(fp_t));
    a1 = (fp_t*)alloc_pinmemory(block_size * k * sizeof(fp_t));
    b1 = (fp_t*)alloc_pinmemory(block_size * k * sizeof(fp_t));
    c1 = (fp_t*)alloc_pinmemory(block_size * block_size * sizeof(fp_t));
    if(a1 == NULL || b1 == NULL || c1 == NULL){
        printf("alloc pin memory  failed\n");
        
    }  
    hipHostRegister(a1, block_size * k * sizeof(fp_t), hipHostRegisterDefault);
    hipHostRegister(b1, block_size * k * sizeof(fp_t), hipHostRegisterDefault);
    hipHostRegister(c1, block_size * block_size * sizeof(fp_t), hipHostRegisterDefault);
    hipMalloc(&a0, block_size * k * sizeof(fp_t));
    hipMalloc(&b0, block_size * k * sizeof(fp_t));
    hipMalloc(&c0, block_size * block_size * sizeof(fp_t));
    int num_reqs = CEIL(block_size * k * sizeof(fp_t), max_io_size);
    u_int64_t *h_reqs = (u_int64_t *)malloc(num_reqs * sizeof(u_int64_t));
    u_int64_t *h_reqs2 = (u_int64_t *)malloc(num_reqs * sizeof(u_int64_t));
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);
    fp_t alpha = 1.0f, beta = 0.0f;
    hipEvent_t start, stop, gemm_start, gemm_stop;
    hipStream_t streama,streamb,streamc;
    hipStreamCreate(&streama);
    hipStreamCreate(&streamb);
    hipStreamCreate(&streamc);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&gemm_start);
    hipEventCreate(&gemm_stop);
    hipEventRecord(start, 0);
    float gemm_ms = 0;
    for (int j = 0; j < n_blocks; j++)
    {
        for (int i = 0; i < num_reqs; i++)
        {
            uint64_t offset = 1ll * i * max_io_size / sizeof(fp_t);
            int row = offset / block_size;
            int col = j * block_size + offset % block_size;
            h_reqs[i] = (b_offset + (1ll * row * n + col) * sizeof(fp_t)) / AEOLUS_LB_SIZE;
        }
        cam_gemm_read(h_reqs,num_reqs,(uintptr_t)b1);
        clear_wait_flag();
        //std::swap(b0, b1);
        hipMemcpyAsync(b0, b1, block_size * k * sizeof(fp_t), hipMemcpyHostToDevice, streamb);
        for (int i = -1; i <= m_blocks+2; i++)
        {
            if (i >= 0 && i < m_blocks)
            {
                // clear_wait_flag();
                hipMemcpyAsync(a0, a1, block_size * k * sizeof(fp_t), hipMemcpyHostToDevice, streama);
                //std::swap(a0, a1);
            }
            if (i  < m_blocks -1)    //read phase
            {
                for (int l = 0; l < num_reqs; l++)
                {
                    uint64_t offset = 1ll * l * max_io_size / sizeof(fp_t);
                    h_reqs[l] = (a_offset + ((i + 1) * block_size * k + offset) * sizeof(fp_t)) / AEOLUS_LB_SIZE;
                }
                cam_gemm_read(h_reqs,num_reqs,(uintptr_t)a1);
                clear_wait_flag();
            }
            if(i>=2 && i<= m_blocks+1){
                // std::swap(c0, c1);
                hipMemcpyAsync(c0, c1, block_size * block_size * sizeof(fp_t), hipMemcpyHostToDevice, streamc);
            }
            if (i >= 3)    //write phase
            {
                // if (i  >= 4)
                // {
                //     clear_wait_flag_write();
                // }
                
                int num_reqs = CEIL(block_size * block_size * sizeof(fp_t), max_io_size);
                for (int l = 0; l < num_reqs; l++)
                {
                    uint64_t offset = 1ll * l * max_io_size / sizeof(fp_t);
                    int row = (i - 1) * block_size + offset / block_size;
                    int col = j * block_size + offset % block_size;
                    h_reqs2[l] = (c_offset + (1ll * row * n + col) * sizeof(fp_t)) / AEOLUS_LB_SIZE;
                    // h_reqs2[l] = l;
                }
                hipStreamSynchronize(streamc);
                cam_gemm_write(h_reqs2,num_reqs,(uintptr_t)c1);
                clear_wait_flag_write();
                
            }
            
            if (i >= 1 && i <= m_blocks)   //gemm compute phase
            {
                hipEventRecord(gemm_start, 0);
                hipStreamSynchronize(streama);
                hipStreamSynchronize(streamb);
                hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, block_size, block_size, k, &alpha, b0, HIP_R_32F, block_size, a0, HIP_R_32F, k, &beta, c0, HIP_R_32F, block_size, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
                hipEventRecord(gemm_stop, 0);
                hipEventSynchronize(gemm_stop);
                float ms;
                hipEventElapsedTime(&ms, gemm_start, gemm_stop);
                gemm_ms += ms;
            }
         }
        // clear_wait_flag_write();
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("m = %d, n = %d, k = %d, block_size = %ld, time = %f ms, tflops = %f\n", m, n, k, block_size, ms, 2.0 * m * n * k / ms / 1e9);
    printf("gemm time = %f ms, num_ssds = %d, max_io_size = %ld, num_queues = %d\n", gemm_ms, num_ssds, max_io_size, num_queues_per_ssd);
    printf("%d %ld %d %ld %f %f %d\n", n, block_size, num_ssds, max_io_size, gemm_ms, ms, num_queues_per_ssd);
    hipblasDestroy(handle);
    free_pinmemory(a1);
    free_pinmemory(b1);
    free_pinmemory(c1);
    hipFree(a0);
    hipFree(b0);
    hipFree(c0);
    hipStreamDestroy(streama);
    hipStreamDestroy(streamb);
    hipStreamDestroy(streamc);
    free(h_reqs);
    cam_clean_up();
    return 0;
}